// In this method, sorting vertices based on in_degree helps to make computation faster
// #include <cuda_stream.h>

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <sys/time.h>
using namespace std;

#define MAX_ITER 1000  // Maximum number of iterations
#define DAMPING_FACTOR 0.85
#define THRESHOLD 1e-6
#define FULL_MASK 0xffffffff

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

__global__ void find_sum(const int start, const int end, const float* arr, const int* col_idx, float* result, const int num_nodes, const int out_degree, int vertex){
    const int index = threadIdx.x + start;
    float value = 0.f;
    const int laneId = threadIdx.x % 32;
    const int n = end - start + 1;
        
    // handle last n%32 elements seperately
    if(threadIdx.x >= n / 32 * 32){
        if(laneId == 0){
            for(int i = index; i <= end; i++) value += arr[col_idx[i]];
        }
    }else{
        value = arr[col_idx[index]];
        for(int offset = 16; offset >= 1; offset /= 2) value += __shfl_down_sync(FULL_MASK, value, offset);
    }
    __shared__ float blockSum;
    if(threadIdx.x == 0)  blockSum = 0.0f;
    __syncthreads();
    // __threadfence_block();
    if(laneId == 0) atomicAdd(&blockSum, value);
    __syncthreads();
    // __threadfence_block();
    if(threadIdx.x == 0) *result = ( (1.0f - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * blockSum) / out_degree;
    // __threadfence_block();
}

__global__ void pageRankKernel(const int *row_ptr, const int *col_idx, const int* out_degree, float *new_contribution, const float *old_contribution, const int num_nodes) {
    const int v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v < num_nodes) {

        const int start = row_ptr[v];
        const int end = row_ptr[v+1] - 1;
        const int in_degree = end - start + 1;
        if(in_degree <= 32*3){
            register float total_contribution = 0.0f;
            // let u = col_idx[j], then u->v is an edge in the graph
            for (int j = start; j <= end; j++) total_contribution += old_contribution[col_idx[j]];
            new_contribution[v] = ( (1.0f - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * total_contribution) / out_degree[v];
       }else if(in_degree <= 1024){
            // computing contribution
            find_sum <<< 1, in_degree>>> (start, end, old_contribution, col_idx, (new_contribution + v), num_nodes, out_degree[v], v);
       }else{
           printf("Currently out of scope\n");
       }
    }
}

__global__ void finalPageRankKernel(const int* row_ptr, const int* col_idx, float* rank, const float* old_contribution, const int num_nodes) {
    const int v = blockIdx.x * blockDim.x + threadIdx.x;
    if(v < num_nodes){
        register float total_contribution = 0.0f;
        for(int j = row_ptr[v]; j < row_ptr[v + 1]; j++) total_contribution += old_contribution[col_idx[j]];
        rank[v] = (1.0f - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * total_contribution;
    }
}

__global__ void initializeContribution (float* contribution, const int* out_degree, const int num_nodes){
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u < num_nodes) contribution[u] = 1.0f / num_nodes / out_degree[u];
}

float* pageRank(const int *row_ptr, const int *col_idx, const int* out_degree, int num_nodes, int num_edges) {
    int num_blocks = (num_nodes + 255) / 256;
    float random_contribution = (1.0f - DAMPING_FACTOR) / num_nodes;
    
    // d_contribution[u] = page_rank value contributed by u to its out neighbours.
    float *d_old_contribution;
    hipMalloc(&d_old_contribution, num_nodes * sizeof(float)); 
    
    // d_out_degree[u] = out_degree[u] stored inside gpu
    int *d_out_degree;
    hipMalloc(&d_out_degree, num_nodes * sizeof(int));
    hipMemcpyAsync(d_out_degree, out_degree, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    
    // intializinf the contribution[u] = (1/n)/(out_degree[u])
    initializeContribution <<<num_blocks, 256>>> (d_old_contribution, d_out_degree, num_nodes);

    // allocating and copying CSR format to GPU
    int *d_row_ptr, *d_col_idx;
    hipMalloc(&d_row_ptr, (num_nodes + 1) * sizeof(int));
    hipMalloc(&d_col_idx, num_edges * sizeof(int));
    hipMemcpyAsync(d_row_ptr, row_ptr, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_col_idx, col_idx, num_edges * sizeof(int), hipMemcpyHostToDevice);
    
    // rank values
    float* d_new_contribution;
    hipMalloc(&d_new_contribution, num_nodes * sizeof(float));

    bool new_flag = true;
    // computing page rank values
    for (int i = 0; i < MAX_ITER - 1; i++) {
        if(new_flag) pageRankKernel <<<num_blocks, 256>>> (d_row_ptr, d_col_idx, d_out_degree, d_new_contribution, d_old_contribution, num_nodes);
        else pageRankKernel <<<num_nodes, 256>>> (d_row_ptr, d_col_idx, d_out_degree, d_old_contribution, d_new_contribution, num_nodes);
        // cudaDeviceSynchronize();
        new_flag = !(new_flag);
    }

    float *rank = (float*)malloc(num_nodes * sizeof(float));
    if(new_flag){
        finalPageRankKernel <<< num_nodes, 256 >>> (d_row_ptr, d_col_idx, d_new_contribution, d_old_contribution, num_nodes);
        hipMemcpy(rank, d_new_contribution, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    }else{
        finalPageRankKernel <<< num_nodes, 256 >>> (d_row_ptr, d_col_idx, d_old_contribution, d_new_contribution, num_nodes);
        hipMemcpy(rank, d_old_contribution, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    }

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_old_contribution);
    hipFree(d_new_contribution);
    hipFree(d_out_degree);

    return rank;
}

void print_page_rank(float* rank, int num_nodes){
    printf("Final page rank values:\n");
    for(int u = 0; u < num_nodes; u++) printf("pageRank[%d] = %f\n", u, rank[u]);
}
// assumptions:
// 1) the graph is unweighted, directed.
// 2) the graph may have multiple edges, self loops.

int main() {
    int num_nodes, num_edges;
    scanf("%d %d", &num_nodes, &num_edges);
    
    vector <vector <int>> in_neighbours(num_nodes);
    int* out_degree = (int*)calloc(num_nodes, sizeof(int));

    for(int edge = 0; edge < num_edges; edge++){
        int u, v;
        scanf("%d %d", &u, &v);
        in_neighbours[v].push_back(u);
        out_degree[u]++;
    }

    int* in_neighbour_index = (int*)malloc((num_nodes + 1)*sizeof(int));  // ROW array in CSR representation
    int* in_neighbour = (int*)malloc(num_edges * sizeof(int));            // COL array in CSR representation

    int edge = 0;
    in_neighbour_index[0] = 0;
    for(int v = 0; v < num_nodes; v++){
        for(int& u: in_neighbours[v]) in_neighbour[edge++] = u;
        in_neighbour_index[v+1] = in_neighbour_index[v] + in_neighbours[v].size();
    }

    /*
    printf("Outdegree:\n");
    for(int u = 0; u < num_nodes; u++) printf("outdegree[%d] = %d\n", u, out_degree[u]);
    printf("\n\n");

    printf("Row array:\n");
    for(int i = 0; i <= num_nodes; i++) printf("%d ", in_neighbour_index[i]);
    printf("\n\n");

    printf("Col array:\n");
    for(int i = 0; i < num_edges; i++) printf("%d ", in_neighbour[i]);
    printf("\n\n");
    */
    
    double t1 = rtclock();
    // Call PageRank function
    float* rank = pageRank(in_neighbour_index, in_neighbour, out_degree, num_nodes, num_edges);
    double t2 = rtclock();

    print_page_rank(rank, num_nodes);
    printf("\nConsumed time: %f\n", t2 - t1);

    return 0;
}
