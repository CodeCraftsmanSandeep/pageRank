#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <sys/time.h>

using namespace std;

#define MAX_ITER 100  // Maximum number of iterations
#define DAMPING_FACTOR 0.85
#define THRESHOLD 1e-6


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}




double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


__global__ void pageRankKernel(const int *row_ptr, const int *col_idx, const float *rank, float *new_rank, const int num_nodes) {
    const int v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v < num_nodes) {
        register float sum = 0.0f;
        
        // u->v is edge in graph
        // u = col_idx[2*j]
        // out_degre = col_idx[2*j + 1]
        for (int j = row_ptr[v]; j < row_ptr[v + 1]; j++) sum += rank[ col_idx[2*j] ] / col_idx[2*j + 1];
        new_rank[v] = (1.0f - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * sum;
    }
}

__global__ void initializePageRank(float* rank, const int num_nodes){
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u < num_nodes) rank[u] = 1.0f / num_nodes;
}

float* pageRank(const int *row_ptr, const int *col_idx, int num_nodes, int num_edges) {
    int num_blocks = (num_nodes + 255) / 256;
    float *d_rank, *d_new_rank;
    int *d_row_ptr, *d_col_idx;

    hipMalloc(&d_rank, num_nodes * sizeof(float)); 
    initializePageRank <<<num_blocks, 256>>> (d_rank, num_nodes);

    hipMalloc(&d_row_ptr, (num_nodes + 1) * sizeof(int));
    hipMalloc(&d_col_idx, 2*num_edges * sizeof(int));
    hipMalloc(&d_new_rank, num_nodes * sizeof(float));

    hipMemcpyAsync(d_row_ptr, row_ptr, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_col_idx, col_idx, 2*num_edges * sizeof(int), hipMemcpyHostToDevice);

    bool is_old = true;
    for (int i = 0; i < MAX_ITER; i++) {
        if(is_old) pageRankKernel<<<num_blocks, 256>>>(d_row_ptr, d_col_idx, d_rank, d_new_rank, num_nodes);
        else  pageRankKernel<<<num_blocks, 256>>>(d_row_ptr, d_col_idx, d_new_rank, d_rank, num_nodes);
        is_old = !(is_old);
    }

    float* rank = (float*)malloc(num_nodes * sizeof(float));
    if(is_old) hipMemcpy(rank, d_rank, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    else hipMemcpy(rank, d_new_rank, num_nodes * sizeof(float), hipMemcpyDeviceToHost); 

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_rank);
    hipFree(d_new_rank);

    return rank;
}

void print_page_rank(float* rank, int num_nodes){
    printf("Final page rank values:\n");
    for(int u = 0; u < num_nodes; u++) printf("pageRank[%d] = %f\n", u, rank[u]);
}

// assumptions:
// 1) the graph is unweighted, directed.
// 2) the graph may have multiple edges, self loops.

struct Node{
    int vertex;
    struct Node* next;

    Node(int vertex): vertex(vertex), next(nullptr) {}  
};

int main() {
    int num_nodes, num_edges;
    scanf("%d %d", &num_nodes, &num_edges);

    Node** in_neighbours = new Node*[num_nodes];
    Node** tail = new Node*[num_nodes];

    for(int u = 0; u < num_nodes; u++){
        tail[u] = in_neighbours[u] = nullptr;
    }

    int* out_degree = (int*)calloc(num_nodes, sizeof(int));

    for(int edge = 0; edge < num_edges; edge++){
        int u, v;
        scanf("%d %d", &u, &v);
        if(tail[v] == nullptr){
            tail[v] = in_neighbours[v] = new Node(u);
        }else{
            tail[v]->next = new Node(u);
            tail[v] = tail[v]->next;
        }
        out_degree[u]++;
    }
    delete[] tail;

    int* in_neighbour_index = new int[num_nodes + 1];  // Row array in CSR format
    int* in_neighbour = new int[2*num_edges];            // Col array in CSR format

    int edge = 0;
    in_neighbour_index[0] = 0;
    for(int v = 0; v < num_nodes; v++){
        int end = in_neighbour_index[v];

        Node* trav = in_neighbours[v];
        while(trav != nullptr){
            in_neighbour[2*(edge)] = trav->vertex;
            in_neighbour[2*(edge) + 1] = out_degree[trav->vertex];
            edge++;
            end++;
            Node* next_ptr = trav->next;
            delete trav;
            trav = next_ptr;
        }
        in_neighbour_index[v+1] = end;
    }
    delete[] in_neighbours;

    double t1 = rtclock();

    // Call PageRank function
    float* rank = pageRank(in_neighbour_index, in_neighbour, num_nodes, num_edges);

    double t2 = rtclock();

    printf("Final page rank values:\n");
    for(int u = 0; u < num_nodes; u++) printf("pageRank[%d] = %f\n", u, rank[u]);

    printf("\nConsumed time: %f\n", t2 - t1);
    return 0;
}
