#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <sys/time.h>
using namespace std;

#define MAX_ITER 1000  // Maximum number of iterations
#define DAMPING_FACTOR 0.85
#define THRESHOLD 1e-5

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

__global__ void pageRankKernel(const int *row_ptr, const int *col_idx, const float *contribution, float *rank, int num_nodes) {
    int v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v < num_nodes) {
        float total_contribution = 0.0f;
        for (int j = row_ptr[v]; j < row_ptr[v + 1]; j++) total_contribution += contribution[col_idx[j]];  // let u = col_idx[j] then u -> v is a edge in the graph
        rank[v] = (1.0f - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * total_contribution;
    }
}

__global__ void initializeContribution (float* contribution, int* out_degree, int num_nodes){
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u < num_nodes) contribution[u] = 1.0f / num_nodes / out_degree[u];
}

__global__ void precompute_contribution(const float* rank, float* contribution, const int* out_degree, int num_nodes){
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u < num_nodes) contribution[u] = rank[u] / out_degree[u];
}

void pageRank(const int *row_ptr, const int *col_idx, const int* out_degree, int num_nodes, int num_edges) {
    int num_blocks = (num_nodes + 255) / 256;
    
    // d_contribution[u] = page_rank value contributed by u to its out neighbours.
    float *d_contribution;
    hipMalloc(&d_contribution, num_nodes * sizeof(float)); 
    
    // d_out_degree[u] = out_degree[u] stored inside gpu
    int *d_out_degree;
    hipMalloc(&d_out_degree, num_nodes * sizeof(int));
    hipMemcpyAsync(d_out_degree, out_degree, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    
    // intializinf the contribution[u] = (1/n)/(out_degree[u])
    initializeContribution <<<num_blocks, 256>>> (d_contribution, d_out_degree, num_nodes);

    // allocating and copying CSR format to GPU
    int *d_row_ptr, *d_col_idx;
    hipMalloc(&d_row_ptr, (num_nodes + 1) * sizeof(int));
    hipMalloc(&d_col_idx, num_edges * sizeof(int));
    hipMemcpyAsync(d_row_ptr, row_ptr, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_col_idx, col_idx, num_edges * sizeof(int), hipMemcpyHostToDevice);
    
    // rank values
    float* d_rank;
    hipMalloc(&d_rank, num_nodes * sizeof(float));

    // computing page rank values
    for (int i = 0; i < MAX_ITER - 1; i++) {
        pageRankKernel <<<num_blocks, 256>>> (d_row_ptr, d_col_idx, d_contribution, d_rank, num_nodes);
        precompute_contribution <<<num_blocks, 256>>> (d_rank, d_contribution, d_out_degree, num_nodes);
    }
    pageRankKernel <<<num_blocks, 256>>> (d_row_ptr, d_col_idx, d_rank, d_contribution, num_nodes);

    float rank[num_nodes];
    hipMemcpy(rank, d_rank, num_nodes * sizeof(float), hipMemcpyDeviceToHost);

    printf("Final page rank values:\n");
    for(int u = 0; u < num_nodes; u++) printf("pageRank[%d] = %f\n", u, rank[u]);

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_rank);
    hipFree(d_contribution);
    hipFree(d_out_degree);
}

// assumptions:
// 1) the graph is unweighted, directed.
// 2) the graph may have multiple edges, self loops.

int main() {
    int num_nodes, num_edges;
    scanf("%d %d", &num_nodes, &num_edges);

    vector <vector <int>> in_neighbours(num_nodes);
    int* out_degree = (int*)calloc(num_nodes, sizeof(int));

    for(int edge = 0; edge < num_edges; edge++){
        int u, v;
        scanf("%d %d", &u, &v);
        in_neighbours[v].push_back(u);
        out_degree[u]++;
    }

    int in_neighbour_index[num_nodes + 1];  // Row array in CSR format
    int in_neighbour[num_edges];            // Col array in CSR format

    int edge = 0;
    in_neighbour_index[0] = 0;
    for(int v = 0; v < num_nodes; v++){
        for(int& u: in_neighbours[v]) in_neighbour[edge++] = u;
        in_neighbour_index[v+1] = in_neighbour_index[v] + in_neighbours[v].size();
    }

    double t1 = rtclock();
    // Call PageRank function
    pageRank(in_neighbour_index, in_neighbour, out_degree, num_nodes, num_edges);
    double t2 = rtclock();
    
    printf("Consumed time: %f\n", t2 - t1);

    return 0;
}