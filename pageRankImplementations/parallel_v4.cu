#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <sys/time.h>
using namespace std;

#define MAX_ITER 1000  // Maximum number of iterations
#define DAMPING_FACTOR 0.85
#define THRESHOLD 1e-5

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

__global__ void pageRankKernel(const int *row_ptr, const int *col_idx, const int* out_degree, float *new_contribution, const float *old_contribution, const int num_nodes) {
    const int v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v < num_nodes) {
        register float total_contribution = 0.0f;
        for (int j = row_ptr[v]; j < row_ptr[v + 1]; j++) total_contribution += old_contribution[col_idx[j]];  // let u = col_idx[j] then u -> v is a edge in the graph
        new_contribution[v] = ( (1.0f - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * total_contribution) / out_degree[v];
    }
}

__global__ void finalPageRankKernel(const int* row_ptr, const int* col_idx, float* rank, const float* old_contribution, const int num_nodes) {
    const int v = blockIdx.x * blockDim.x + threadIdx.x;
    if(v < num_nodes){
        register float total_contribution = 0.0f;
        for(int j = row_ptr[v]; j < row_ptr[v + 1]; j++) total_contribution += old_contribution[col_idx[j]];
        rank[v] = (1.0f - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * total_contribution;
    }
}

__global__ void initializeContribution (float* contribution, const int* out_degree, const int num_nodes){
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u < num_nodes) contribution[u] = 1.0f / num_nodes / out_degree[u];
}

void pageRank(const int *row_ptr, const int *col_idx, const int* out_degree, int num_nodes, int num_edges) {
    int num_blocks = (num_nodes + 255) / 256;
    float random_contribution = (1.0f - DAMPING_FACTOR) / num_nodes;
    
    // d_contribution[u] = page_rank value contributed by u to its out neighbours.
    float *d_old_contribution;
    hipMalloc(&d_old_contribution, num_nodes * sizeof(float)); 
    
    // d_out_degree[u] = out_degree[u] stored inside gpu
    int *d_out_degree;
    hipMalloc(&d_out_degree, num_nodes * sizeof(int));
    hipMemcpyAsync(d_out_degree, out_degree, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    
    // intializinf the contribution[u] = (1/n)/(out_degree[u])
    initializeContribution <<<num_blocks, 256>>> (d_old_contribution, d_out_degree, num_nodes);

    // allocating and copying CSR format to GPU
    int *d_row_ptr, *d_col_idx;
    hipMalloc(&d_row_ptr, (num_nodes + 1) * sizeof(int));
    hipMalloc(&d_col_idx, num_edges * sizeof(int));
    hipMemcpyAsync(d_row_ptr, row_ptr, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_col_idx, col_idx, num_edges * sizeof(int), hipMemcpyHostToDevice);
    
    // rank values
    float* d_new_contribution;
    hipMalloc(&d_new_contribution, num_nodes * sizeof(float));

    bool new_flag = true;
    // computing page rank values
    for (int i = 0; i < MAX_ITER - 1; i++) {
        if(new_flag) pageRankKernel <<<num_blocks, 256>>> (d_row_ptr, d_col_idx, d_out_degree, d_new_contribution, d_old_contribution, num_nodes);
        else pageRankKernel <<<num_nodes, 256>>> (d_row_ptr, d_col_idx, d_out_degree, d_old_contribution, d_new_contribution, num_nodes);

        new_flag = !(new_flag);
    }

    float rank[num_nodes];
    if(new_flag){
        finalPageRankKernel <<< num_nodes, 256 >>> (d_row_ptr, d_col_idx, d_new_contribution, d_old_contribution, num_nodes);
        hipMemcpy(rank, d_new_contribution, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    }else{
        finalPageRankKernel <<< num_nodes, 256 >>> (d_row_ptr, d_col_idx, d_old_contribution, d_new_contribution, num_nodes);
        hipMemcpy(rank, d_old_contribution, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    }

    printf("Final page rank values:\n");
    for(int u = 0; u < num_nodes; u++) printf("pageRank[%d] = %f\n", u, rank[u]);

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_old_contribution);
    hipFree(d_new_contribution);
    hipFree(d_out_degree);
}

// assumptions:
// 1) the graph is unweighted, directed.
// 2) the graph may have multiple edges, self loops.

int main() {
    int num_nodes, num_edges;
    scanf("%d %d", &num_nodes, &num_edges);

    vector <vector <int>> in_neighbours(num_nodes);
    int* out_degree = (int*)calloc(num_nodes, sizeof(int));

    for(int edge = 0; edge < num_edges; edge++){
        int u, v;
        scanf("%d %d", &u, &v);
        in_neighbours[v].push_back(u);
        out_degree[u]++;
    }

    int in_neighbour_index[num_nodes + 1];  // Row array in CSR format
    int in_neighbour[num_edges];            // Col array in CSR format

    int edge = 0;
    in_neighbour_index[0] = 0;
    for(int v = 0; v < num_nodes; v++){
        for(int& u: in_neighbours[v]) in_neighbour[edge++] = u;
        in_neighbour_index[v+1] = in_neighbour_index[v] + in_neighbours[v].size();
    }

    double t1 = rtclock();
    // Call PageRank function
    pageRank(in_neighbour_index, in_neighbour, out_degree, num_nodes, num_edges);
    double t2 = rtclock();
    
    printf("Consumed time: %f\n", t2 - t1);

    return 0;
}